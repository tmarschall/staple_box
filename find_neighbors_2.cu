#include "hip/hip_runtime.h"
// -*- c++ -*-
/*
* find_neighbors.cu
*
*  
*
*
*/

#include <hip/hip_runtime.h>
#include "staple_box.h"
#include "cudaErr.h"
#include "data_primitives.h"
#include <sm_20_atomic_functions.h>
#include <math.h>

using namespace std;


///////////////////////////////////////////////////////////////
// Find the Cell ID for each particle:
//  The list of cell IDs for each particle is returned to pnCellID
//  A list of which particles are in each cell is returned to pnCellList
//
// *NOTE* if there are more than nMaxPPC particles in a given cell,
//  not all of these particles will get added to the cell list
///////////////////////////////////////////////////////////////
__global__ void find_cells(int nStaples, int nMaxPPC, double dCellW, double dCellH,
			   int nCellCols, double dL, double *pdX, double *pdY, 
			   int *pnCellID, int *pnPPC, int *pnCellList)
{
  // Assign each thread a unique ID accross all thread-blocks, this is its particle ID
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nStaples) {
    double dX = pdX[nPID];
    double dY = pdY[nPID];
    
    // I often allow the stored coordinates to drift slightly outside the box limits
    //  until 
    if (dY > dL)
      {
	dY -= dL;
	pdY[nPID] = dY;
      }
    else if (dY < 0)
      {
	dY += dL;
	pdY[nPID] = dY;
      }
    if (dX > dL)
      {
	dX -= dL;
	pdX[nPID] = dX;
      }
    else if (dX < 0)
      {
	dX += dL;
	pdX[nPID] = dX;
      }

    //find the cell ID, add a particle to that cell 
    int nCol = (int)(dX / dCellW);
    int nRow = (int)(dY / dCellH); 
    int nCellID = nCol + nRow * nCellCols;
    pnCellID[nPID] = nCellID;

    // Add 1 particle to a cell safely (only allows one thread to access the memory
    //  address at a time). nPPC is the original value, not the result of addition 
    int nPPC = atomicAdd(pnPPC + nCellID, 1);
    
    // only add particle to cell if there is not already the maximum number in cell
    if (nPPC < nMaxPPC)
      pnCellList[nCellID * nMaxPPC + nPPC] = nPID;
    else
      nPPC = atomicAdd(pnPPC + nCellID, -1);

    nPID += nThreads;
  }
}


////////////////////////////////////////////////////////////////
// Here a list of possible contacts is created for each particle
//  The list of neighbors is returned to pnNbrList
//
// This is one function that I may target for optimization in
//  the future because I know it is slowed down by branch divergence
////////////////////////////////////////////////////////////////
__global__ void find_nbrs(int nStaples, int nMaxPPC, int *pnCellID, int *pnPPC, 
			  int *pnCellList, int *pnAdjCells, int nMaxNbrs, int *pnNPP, 
			  int *pnNbrList, double *pdX, double *pdY, double *pdR, 
			  double dEpsilon, double dL, double dGamma)
{
  extern __shared__ int sData[];
  int thid = threadIdx.x;
  int blsz = blockDim.x;
  int blid = blockIdx.x;
  int nPID = thid + blid * blsz;
  int nThreads = gridDim.x * blsz;

  while (nPID < nStaples)
    {
      double dX = pdX[nPID];
      double dY = pdY[nPID];
      double dR = pdR[nPID];
      int nNbrs = 0;

      // Particles in adjacent cells are added if they are close enough to 
      //  interact without each moving by more than dEpsilon/2
      int nCellID = pnCellID[nPID];
      int nP = pnPPC[nCellID];
      for (int p = 0; p < nP; p++)
	{
	  int nAdjPID = pnCellList[nCellID*nMaxPPC + p];
	  if (nAdjPID != nPID)
	    {
	      double dSigma = dR + pdR[nAdjPID] + dEpsilon;
	      double dDeltaY = dY - pdY[nAdjPID];
	      dDeltaY += dL * ((dDeltaY < -0.5 * dL) - (dDeltaY > 0.5 * dL));
	      
	      if (fabs(dDeltaY) < dSigma)
		{
		  double dDeltaX = dX - pdX[nAdjPID];
		  dDeltaX += dL * ((dDeltaX < -0.5 * dL) - (dDeltaX > 0.5 * dL));
		  double dDeltaRx = dDeltaX + dGamma * dDeltaY;
		  double dDeltaRx2 = dDeltaX + 0.5 * dDeltaY;
		  if (fabs(dDeltaRx) < dSigma || fabs(dDeltaRx2) < dSigma)
		    {
		      // This indexing makes global memory accesses more coalesced
		      if (nNbrs < nMaxNbrs)
			{
			  //pnNbrList[nStaples * nNbrs + nPID] = nAdjPID;
			  sData[blsz * nNbrs + thid] = nAdjPID;
			  nNbrs += 1;
			}
		    }
		}
	    }
	}

      for (int nc = 0; nc < 8; nc++)
	{
	  int nAdjCID = pnAdjCells[8 * nCellID + nc];
	  nP = pnPPC[nAdjCID];
	  for (int p = 0; p < nP; p++)
	    {
	      int nAdjPID = pnCellList[nAdjCID*nMaxPPC + p];
	      
	      // The maximum distance at which two particles could contact
	      //  plus a little bit of moving room - dEpsilon 
	      double dSigma = dR + pdR[nAdjPID] + dEpsilon;
	      double dDeltaY = dY - pdY[nAdjPID];

	      // Make sure were finding the closest separation
	      dDeltaY += dL * ((dDeltaY < -0.5 * dL) - (dDeltaY > 0.5 * dL));

	      if (fabs(dDeltaY) < dSigma)
		{
		  double dDeltaX = dX - pdX[nAdjPID];
		  dDeltaX += dL * ((dDeltaX < -0.5 * dL) - (dDeltaX > 0.5 * dL));

		  // Go to unsheared coordinates
		  double dDeltaRx = dDeltaX + dGamma * dDeltaY;
		  // Also look at distance when the strain parameter is at its max (0.5)
		  double dDeltaRx2 = dDeltaX + 0.5 * dDeltaY;
		  if (fabs(dDeltaRx) < dSigma || fabs(dDeltaRx2) < dSigma)
		    {
		      if (nNbrs < nMaxNbrs)
			{
			  //pnNbrList[nStaples * nNbrs + nPID] = nAdjPID;
			  sData[blsz * nNbrs + thid] = nAdjPID;
			  nNbrs += 1;
			}
		    }
		}
	    }
	  
	}
      pnNPP[nPID] = nNbrs;
      for (int n = 0; n < nNbrs; n++) {
	pnNbrList[nStaples * n + nPID] = sData[blsz * n + thid];
      }

      nPID += nThreads;
    }
}


__global__ void get_nbr_blocks(int nStaples, int *pnNNbrs, int *pnNbrList, int *pnBlockNbrs, int *pnBlockList)
{
  extern __shared__ int sData[];
  int thid = threadIdx.x;
  int blid = blockIdx.x;
  int blsz = blockDim.x;
  int nPID = thid + blid * blsz;
  
  while (nPID < nStaples) {
    sData[thid] = pnNNbrs[nPID];
    int nNbrs = sData[thid];
    __syncthreads();

    int offset = 1;
    for (int d = blsz / 2; d > 0; d /= 2) {
      if (thid < d) {
	int ai = offset*(2*thid + 1) - 1;
	int bi = offset*(2*thid + 2) - 1;
	sData[bi] += sData[ai];
      }
      offset *= 2;
      __syncthreads();
    }
    
    int nTotNbrs = sData[blsz - 1];
    int nSortMax = 256;
    int s = nTotNbrs / 256;
    while (s > 0) {
      nSortMax *= 2;
      s /= 2;
    }
    __syncthreads();
    if (thid == 0) {
      sData[blsz - 1] = 0;
      //pnBlockNbrs[blid] = nTotNbrs;
    }
    
    for (int d = 1; d < blsz; d *= 2) {
      __syncthreads();
      offset /= 2;
      if (thid < d) {
	int ai = offset*(2*thid + 1) - 1;
	int bi = offset*(2*thid + 2) - 1;
	int temp = sData[ai];
	sData[ai] = sData[bi];
	sData[bi] += temp;
      }
    }
    
    __syncthreads();
    //int *pnList = &sData[blsz];
    int nID = sData[thid];
    for (int n = 0; n < nNbrs; n++) {
	sData[blsz + nID + n] = pnNbrList[n * nStaples + nPID];
    }
    for (int t = nTotNbrs + thid; t < nSortMax; t += blsz) {
      sData[blsz + t] = nStaples;
    }
    
    __syncthreads();
    //for (int t = thid; t < 2048; t += blsz){
    //pnBlockList[2048*blid + t] = pnList[t];
    //}
    
    for (int oblock = 1; oblock < nSortMax; oblock *= 2) {
      int t = thid;
      int obid = t / oblock;
      while (obid < nSortMax / (2 * oblock)) {
	for (int iblock = oblock; iblock > 0; iblock /= 2) {
	  int s = (t % oblock);
	  int ibid = s / iblock;
	  while (ibid < oblock / iblock) {
	    int ai, bi;
	    if (obid % 2) {
	      bi = blsz + 2 * (oblock * obid + iblock * ibid) + s % iblock;
	      ai = bi + iblock;
	    }
	    else {
	      ai = blsz + 2 * (oblock * obid + iblock * ibid) + s % iblock;
	      bi = ai + iblock;
	    }
	    if (sData[ai] > sData[bi]) {
	      int temp = sData[ai];
	      sData[ai] = sData[bi];
	      sData[bi] = temp;
	    }
	    s += blsz;
	    ibid = s / iblock;
	  }
	  if (iblock > 32)
	    __syncthreads();
	}
	t += blsz;
	obid = t / oblock;
      }
      __syncthreads();
    }

    
    //int *pnCID = &pnList[nSortMax];
    for (int t = thid; t < nSortMax - 1; t += blsz) {
      if (sData[blsz + t] == sData[blsz + t + 1]) 
	sData[blsz + nSortMax + t] = 0;
      else
	sData[blsz + nSortMax + t] = 1;
    }
    if (thid == 0)
      sData[blsz + 2 * nSortMax - 1] = 0;
    __syncthreads();

    offset = 1;
    for (int d = nSortMax / 2; d > 0; d /= 2) {
      int t = thid;
      while (t < d) {
	int ai = blsz + nSortMax + offset*(2*t + 1) - 1;
	int bi = blsz + nSortMax + offset*(2*t + 2) - 1;
	sData[bi] += sData[ai];
	t += blsz;
      }
      offset *= 2;
      __syncthreads();
    }
    
    nTotNbrs = sData[blsz + 2 * nSortMax - 1];
    __syncthreads();
    if (thid == 0) {
      sData[blsz + 2*nSortMax - 1] = 0;
      pnBlockNbrs[blid] = nTotNbrs;
    }
    
    for (int d = 1; d < nSortMax; d *= 2) {
      __syncthreads();
      int t = thid;
      offset /= 2;
      while (t < d) {
	int ai = blsz + nSortMax + offset*(2*t + 1) - 1;
	int bi = blsz + nSortMax + offset*(2*t + 2) - 1;
	int temp = sData[ai];
	sData[ai] = sData[bi];
	sData[bi] += temp;
	t += blsz;
      }
    }

    __syncthreads();
    for (int t = thid; t < nSortMax - 1; t += blsz) {
      if (sData[blsz + t] != sData[blsz + t + 1]) 
	pnBlockList[blid*8*blsz + sData[blsz + nSortMax + t]] = sData[blsz + t];
    }
    __syncthreads();
    for (int t = thid; t < nTotNbrs; t += blsz) {
      sData[t] = pnBlockList[blid*8*blsz + t];
    }
    __syncthreads();

    for (int n = 0; n < nNbrs; n++) {
      int oldID = pnNbrList[nStaples * n + nPID];
      int newID = nTotNbrs / 2;
      int diff = max(-newID, min(oldID - sData[newID], nTotNbrs - 1 - newID));
      int dStep = newID / 2;
      newID += diff;
      while (dStep > 0) {
	diff = max(max(-dStep, -newID), min(oldID - sData[newID], min(dStep, nTotNbrs - 1 - newID)));
	newID += diff;
	dStep /= 2;
      }
      pnNbrList[nStaples*n + nPID] = newID;
    }
    
    blid += gridDim.x;
    nPID += gridDim.x * blsz;
  }
    
}

///////////////////////////////////////////////////////////////
// Finds a list of possible contacts for each particle
//
// Usually when things are moving I keep track of an Xmoved and Ymoved
//  and only call this to make a new list of neighbors if some particle
//  has moved more than (dEpsilon / 2) in some direction
///////////////////////////////////////////////////////////////
void Staple_Box::find_neighbors()
{
  // reset each byte to 0
  hipMemset((void *) d_pnPPC, 0, sizeof(int)*m_nCells);
  hipMemset((void *) d_pdXMoved, 0, sizeof(double)*m_nStaples);
  hipMemset((void *) d_pdYMoved, 0, sizeof(double)*m_nStaples);
  hipMemset((void *) d_bNewNbrs, 0, sizeof(int));

  find_cells <<<m_nGridSize, m_nBlockSize>>>
    (m_nStaples, m_nMaxPPC, m_dCellW, m_dCellH, m_nCellCols, 
     m_dL, d_pdX, d_pdY, d_pnCellID, d_pnPPC, d_pnCellList);
  hipDeviceSynchronize();
  checkCudaError("Finding cells");

  find_nbrs <<<m_nGridSize, m_nBlockSize, m_nSM_FindCells>>>
    (m_nStaples, m_nMaxPPC, d_pnCellID, d_pnPPC, d_pnCellList, d_pnAdjCells, 
     m_nMaxNbrs, d_pnNPP, d_pnNbrList, d_pdX, d_pdY, d_pdR, m_dEpsilon, m_dL, m_dGamma);
  hipDeviceSynchronize();
  checkCudaError("Finding neighbors");

  get_nbr_blocks <<<m_nGridSize, m_nBlockSize, m_nSM_GetNbrBlks>>>
    (m_nStaples, d_pnNPP, d_pnNbrList, d_pnBlockNNbrs, d_pnBlockList);
  hipDeviceSynchronize();
  checkCudaError("Getting neighbor blocks");

}


////////////////////////////////////////////////////////////////////////////////////
// Sets gamma back by 1 (used when gamma > 0.5)
//  also finds the cells in the process
//
///////////////////////////////////////////////////////////////////////////////////
__global__ void set_back_coords(int nStaples, int nMaxPPC, double dCellW, double dCellH,
				int nCellCols, double dL, double *pdX, double *pdY, 
				int *pnCellID, int *pnPPC, int *pnCellList)
{
  // Assign each thread a unique ID accross all thread-blocks, this is its particle ID
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nStaples) {
    double dX = pdX[nPID];
    double dY = pdY[nPID];
    
    // I often allow the stored coordinates to drift slightly outside the box limits
    //  until 
    if (dY > dL)
      {
	dY -= dL;
	pdY[nPID] = dY;
      }
    else if (dY < 0)
      {
	dY += dL;
	pdY[nPID] = dY;
      }
    
    // When gamma -> gamma-1, Xi -> Xi + Yi
    dX += dY;
    if (dX < 0)
      {
	dX += dL;
      }
    while (dX > dL)
      {
	dX -= dL;
      }
    pdX[nPID] = dX;


    //find the cell ID, add a particle to that cell 
    int nCol = (int)(dX / dCellW);
    int nRow = (int)(dY / dCellH); 
    int nCellID = nCol + nRow * nCellCols;
    pnCellID[nPID] = nCellID;

    // Add 1 particle to a cell safely (only allows one thread to access the memory
    //  address at a time). nPPC is the original value, not the result of addition 
    int nPPC = atomicAdd(pnPPC + nCellID, 1);
    
    // only add particle to cell if there is not already the maximum number in cell
    if (nPPC < nMaxPPC)
      pnCellList[nCellID * nMaxPPC + nPPC] = nPID;
    else
      nPPC = atomicAdd(pnPPC + nCellID, -1);

    nPID += nThreads;
  }

}

void Staple_Box::set_back_gamma()
{
  hipMemset((void *) d_pnPPC, 0, sizeof(int)*m_nCells);
  hipMemset((void *) d_pdXMoved, 0, sizeof(double)*m_nStaples);
  hipMemset((void *) d_pdYMoved, 0, sizeof(double)*m_nStaples);
  hipMemset((void *) d_bNewNbrs, 0, sizeof(int));

  set_back_coords <<<m_nGridSize, m_nBlockSize>>>
    (m_nStaples, m_nMaxPPC, m_dCellW, m_dCellH, m_nCellCols, 
     m_dL, d_pdX, d_pdY, d_pnCellID, d_pnPPC, d_pnCellList);
  hipDeviceSynchronize();
  checkCudaError("Finding new coordinates, cells");
  m_dGamma -= 1;

  find_nbrs <<<m_nGridSize, m_nBlockSize, m_nSM_FindCells>>>
    (m_nStaples, m_nMaxPPC, d_pnCellID, d_pnPPC, d_pnCellList, d_pnAdjCells, 
     m_nMaxNbrs, d_pnNPP, d_pnNbrList, d_pdX, d_pdY, d_pdR, m_dEpsilon, m_dL, m_dGamma);
  hipDeviceSynchronize();
  checkCudaError("Finding neighbors");
}


////////////////////////////////////////////////////////////////////////////
// Finds cells for all particles regardless of maximum particle per cell
//  used for reordering particles
/////////////////////////////////////////////////////////////////////////
__global__ void find_cells_nomax(int nStaples, double dCellW, double dCellH,
				 int nCellCols, double dL, double *pdX, double *pdY, 
				 int *pnCellID, int *pnPPC)
{
  // Assign each thread a unique ID accross all thread-blocks, this is its particle ID
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nStaples) {
    double dX = pdX[nPID];
    double dY = pdY[nPID];
    
    // Particles are allowed to drift slightly outside the box limits
    //  until cells are reassigned due to a particle drift of dEpsilon/2 
    if (dY > dL) {
      dY -= dL; 
      pdY[nPID] = dY; }
    else if (dY < 0) {
      dY += dL;
      pdY[nPID] = dY; }
    if (dX > dL) {
      dX -= dL; 
      pdX[nPID] = dX; }
    else if (dX < 0) {
      dX += dL;
      pdX[nPID] = dX; }

    //find the cell ID, add a particle to that cell 
    int nCol = (int)(dX / dCellW);
    int nRow = (int)(dY / dCellH); 
    int nCellID = nCol + nRow * nCellCols;
    
    pnCellID[nPID] = nCellID;
    int nPPC = atomicAdd(pnPPC + nCellID, 1);
    
    nPID += nThreads; }
}

__global__ void reorder_part(int nStaples, double *pdTempX, double *pdTempY, 
			     double *pdTempR, int *pnInitID, double *pdX, 
			     double *pdY, double *pdR, int *pnMemID, 
			     int *pnCellID, int *pnCellSID)
{
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nStaples) {
    double dX = pdTempX[nPID];
    double dY = pdTempY[nPID];
    double dR = pdTempR[nPID];
    int nInitID = pnInitID[nPID];

    int nCellID = pnCellID[nPID];
    int nNewID = atomicAdd(pnCellSID + nCellID, 1);
    
    pdX[nNewID] = dX;
    pdY[nNewID] = dY;
    pdR[nNewID] = dR;
    pnMemID[nNewID] = nInitID;

    nPID += nThreads; }
}

__global__ void invert_IDs(int nIDs, int *pnIn, int *pnOut)
{
  int thid = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (thid < nIDs) {
    int i = pnIn[thid];
    pnOut[i] = thid; 
    thid += nThreads; }
    
}

void Staple_Box::reorder_particles()
{
  hipMemset((void *) d_pnPPC, 0, sizeof(int)*m_nCells);

  //find particle cell IDs and number of particles in each cell
  find_cells_nomax <<<m_nGridSize, m_nBlockSize>>>
    (m_nStaples, m_dCellW, m_dCellH, m_nCellCols, 
     m_dL, d_pdX, d_pdY, d_pnCellID, d_pnPPC);
  hipDeviceSynchronize();
  checkCudaError("Reordering particles: Finding cells");

  int *d_pnCellSID;
  double *d_pdTempR;
  hipMalloc((void **) &d_pnCellSID, sizeof(int) * m_nCells);
  hipMalloc((void **) &d_pdTempR, sizeof(double) * m_nStaples);
  hipMemcpy(d_pdTempX, d_pdX, sizeof(double) * m_nStaples, hipMemcpyDeviceToDevice);
  hipMemcpy(d_pdTempY, d_pdY, sizeof(double) * m_nStaples, hipMemcpyDeviceToDevice);
  hipMemcpy(d_pdTempR, d_pdR, sizeof(double) * m_nStaples, hipMemcpyDeviceToDevice);

  exclusive_scan(d_pnPPC, d_pnCellSID, m_nCells);

  /*
  int *h_pnCellSID = (int*) malloc(m_nCells * sizeof(int));
  int *h_pnCellNPart = (int*) malloc(m_nCells * sizeof(int));
  hipMemcpy(h_pnCellNPart, d_pnCellNPart, sizeof(int)*m_nCells, hipMemcpyDeviceToHost);
  hipMemcpy(h_pnCellSID, d_pnCellSID, sizeof(int)*m_nCells, hipMemcpyDeviceToHost);
  for (int c = 0; c < m_nCells; c++)
    {
      printf("%d %d\n", h_pnCellNPart[c], h_pnCellSID[c]);
    }
  free(h_pnCellSID);
  free(h_pnCellNPart);
  */

  //reorder particles based on cell ID (first by Y direction)
  reorder_part <<<m_nGridSize, m_nBlockSize>>>
    (m_nStaples, d_pdTempX, d_pdTempY, d_pdTempR, d_pnInitID, 
     d_pdX, d_pdY, d_pdR, d_pnMemID, d_pnCellID, d_pnCellSID);
  hipDeviceSynchronize();
  checkCudaError("Reordering particles: changing order");

  invert_IDs <<<m_nGridSize, m_nBlockSize>>> (m_nStaples, d_pnMemID, d_pnInitID);
  hipMemcpyAsync(h_pnMemID, d_pnMemID, m_nStaples*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync(h_pdR, d_pdR, m_nStaples*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_pnCellSID);
  hipFree(d_pdTempR);

  find_neighbors();
}


////////////////////////////////////////////////////////////////////////
// Sets the particle IDs to their order in memory
//  so the current IDs become the initial IDs
/////////////////////////////////////////////////////////////////////
void Staple_Box::reset_IDs()
{
  ordered_array(d_pnInitID, m_nStaples, m_nGridSize, m_nBlockSize);
  hipMemcpy(d_pnMemID, d_pnInitID, sizeof(int)*m_nStaples, hipMemcpyDeviceToDevice);
  hipMemcpy(h_pnMemID, d_pnInitID, sizeof(int)*m_nStaples, hipMemcpyDeviceToHost);
}
