#include "hip/hip_runtime.h"
// -*- c++ -*-
/*
*  data_primitives.cu
*
*  kernels for primitive data operations used in several functions
*
*
*/

#include "data_primitives.h"
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#include "cudaErr.h"
#include <stdio.h>


__global__ void block_scan(int nLength, int nBlockSize, int *pnIn, int *pnOut, int nBlocks, int *pnSums)
{
  extern __shared__ int sMem[];
  int thid = threadIdx.x;
  int blid = blockIdx.x;

  while (blid < nBlocks)
    {
      int offset = 1;

      int globalId = thid + blid * nBlockSize;
      int sharedId = thid + thid / 32;
      if (globalId < nLength)
	{
	  sMem[sharedId] = pnIn[globalId];
	  globalId += nBlockSize / 2;
	  sharedId = thid + nBlockSize / 2;
	  sharedId += sharedId / 32;
	  if (globalId < nLength)
	    sMem[sharedId] = pnIn[globalId];
	  else
	    sMem[sharedId] = 0;
	}
      else
	{
	  sMem[sharedId] = 0;
	  sharedId = thid + nBlockSize / 2;
	  sharedId += sharedId / 32;
	  sMem[sharedId] = 0;
	}

      for (int d = nBlockSize / 2; d > 0; d /= 2)
	{
	  __syncthreads();
	  if (thid < d)
	    {
	      int ai = offset*(2*thid + 1) - 1;
	      int bi = offset*(2*thid + 2) - 1;
	      ai += ai / 32;
	      bi += bi / 32;
	      sMem[bi] += sMem[ai];
	    }
	  offset *= 2;
	}
      
      __syncthreads();
      int nSID = thid + blid + 1;
      int nLastID = nBlockSize - 1;
      nLastID += nLastID / 32;
      if (nSID < nBlocks)
	int nPartSum = atomicAdd(pnSums+nSID, sMem[nLastID]);
      __syncthreads();
      if (thid == 0)
	sMem[nLastID] = 0;

      for (int d = 1; d < nBlockSize; d *= 2)
	{
	  offset /= 2;
	  __syncthreads();
	  if (thid < d)
	    {
	      int ai = offset*(2*thid + 1) - 1;
	      int bi = offset*(2*thid + 2) - 1;
	      ai += ai / 32;
	      bi += bi / 32;
	      int temp = sMem[ai];
	      sMem[ai] = sMem[bi];
	      sMem[bi] += temp;
	    }
	}
      __syncthreads();
      
      globalId = blid * nBlockSize + thid;
      sharedId = thid + thid / 32;
      pnOut[globalId] = sMem[sharedId];
      sharedId = thid + nBlockSize / 2;
      sharedId += sharedId / 32;
      pnOut[globalId + nBlockSize / 2] = sMem[sharedId];

      blid += gridDim.x;
      }
}

__global__ void finish_scan(int nLength, int nBlockSize, int *pnOut, int nBlocks, int *pnSums)
{
  int blid = blockIdx.x;
  
  while (blid < nBlocks)
    {
      int globalId = threadIdx.x + blid * nBlockSize;
      if (globalId < nLength)
	{
	  pnOut[globalId] += pnSums[blid];
	  globalId += nBlockSize / 2;
	  if (globalId < nLength)
	    pnOut[globalId] += pnSums[blid];
	}
      
      blid += gridDim.x;
    }
}

void exclusive_scan(int *d_pnIn, int *d_pnOut, int nSize)
{
  int nBlockSize = 512;
  int nBlocks = nSize / nBlockSize + ((nSize % nBlockSize != 0) ? 1 : 0);
  int *d_pnSums;
  hipMalloc((void **) &d_pnSums, sizeof(int) * nBlocks);
  hipMemset((void *) d_pnSums, 0, sizeof(int) * nBlocks);
  
  int nGridDim = ((nBlocks <= 16) ? nBlocks : 16);
  int nBlockDim = nBlockSize / 2;
  int sMemSize = (nBlockSize + nBlockSize / 32) * sizeof(int);
  //printf("Scanning %d x %d with %d bytes smem\n", nGridDim, nBlockDim, sMemSize);

  block_scan <<<nGridDim, nBlockDim, sMemSize>>> (nSize, nBlockSize, d_pnIn, d_pnOut, nBlocks, d_pnSums);
  hipDeviceSynchronize();
  checkCudaError("Performing exclusive scan on blocks");

  if (nBlocks > 1)
    {
      //int *h_pnSums = (int*) malloc(sizeof(int) * nBlocks);
      finish_scan <<<nGridDim, nBlockDim>>> (nSize, nBlockSize, d_pnOut, nBlocks, d_pnSums);
      hipDeviceSynchronize();
      checkCudaError("Adding block sums to block scans");

      //hipMemcpy(h_pnSums, d_pnSums, sizeof(int) * nBlocks)
    }

  hipFree(d_pnSums);
}


__global__ void ordered_arr(int *pnArr, int nSize)
{
  int thid = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;
  
  while (thid < nSize)
    {
      pnArr[thid] = thid;
      thid += nThreads;
    }
}


void ordered_array(int *pnArray, int nSize, int gridSize, int blockSize)
{
  int nBlockSize;
  int nGridSize;
  if (gridSize == 0 || blockSize == 0)
    {
      if (nSize > 16 * 128)
	nBlockSize = 256;
      else
	nBlockSize = 128;
      nGridSize = nSize / nBlockSize + ((nSize % nBlockSize != 0) ? 1 : 0);
    }
  else
    {
      nGridSize = gridSize;
      nBlockSize = blockSize;
    }

  ordered_arr <<<nGridSize, nBlockSize>>> (pnArray, nSize);
  hipDeviceSynchronize();
  checkCudaError("Creating ordered array");
}
